#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return (double) tp.tv_sec + (double)tp.tv_usec*1e-6;
}

#define CLOCK_RATE 1695000  /* Modify from below */
__device__ void sleep(float t) {    
    clock_t t0 = clock64();
    clock_t t1 = t0;
    while ((t1 - t0)/(CLOCK_RATE*1000.0f) < t)
        t1 = clock64();
}

__global__ void mykernel() {
    sleep(1.0);    
}

int main(int argc, char* argv[]) {
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0); 
    clock_t clock_rate = prop.clockRate;

    int num_blocks = atoi(argv[1]);

    dim3 block(1);
    dim3 grid(num_blocks);  /* N blocks */

    double start = cpuSecond();
    mykernel<<<grid,block>>>();
    hipDeviceSynchronize();
    double etime = cpuSecond() - start;

    printf("clock_rate          %10d\n", clock_rate);
    printf("time                %10.2f\n", etime);

    hipDeviceReset();
}
